
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/time.h>

#define NN (1<<10) // number of seeds
#define MM (1<<20) // number of samples per seed
#define THREADBLOCKSIZE 1024
#define LENGTH (N*sizeof(float))
#define INDEX (blockIdx.x * blockDim.x + threadIdx.x)
#define MARK_TIME(t) gettimeofday(&t, NULL)
#define CALC_TIME(t1, t2) (1.0e6 * (t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec))/(1.0e6)
#define D2H hipMemcpyDeviceToHost
#define H2D hipMemcpyHostToDevice

extern __shared__ float sdat[];

/*
Calculates an estimate for pi for each thread. Uses m random numbers
generated using the globalState random number generator
*/
__device__ void d_gen(hiprandState *globalState, int m) {
  int i = INDEX;
  hiprandState localState = globalState[i];

  int spt = m; //number of samples per thread
  int tid = threadIdx.x;
  int count = 0;
  for (int s = 0; s < spt; s++) {
    float rx = hiprand_uniform(&localState);
    float ry = hiprand_uniform(&localState);
    float mag = rx*rx + ry*ry;
    if (mag <= 1.0f) {
      count += 1;
      sdat[tid] += 1.0;
    }
  }

  globalState[i] = localState;
  sdat[tid] *= 1.0/spt;
}

/*
Creates an estimate of pi for an entire block. Sums the
individual thread estimates and then divides by the number of threads
in a block.
*/
__device__ void d_count(float *sums) {
  int tid = threadIdx.x;
  for (int i = blockDim.x/2; i > 0; i >>= 1) {
    if (tid < i)
      sdat[tid] += sdat[tid + i];
   __syncthreads();
  }
  __syncthreads();
  if (tid == 0) {
    sdat[0] *= 1.0/blockDim.x;
    sums[blockIdx.x] = sdat[0];
  }
}

/*
Actuallyt generates the estimates for pi for each block once the random
number generators are correctly set up.
*/
__global__ void generate(hiprandState *globalState, float *sums, int m) {
  d_gen(globalState, m);
  __syncthreads();
  d_count(sums);
  __syncthreads();
}

/*
Sets up the random number generators for the blocks. This step must be
called before the pi estimates are generated.
*/
__global__ void kernel_setup(hiprandState *states) {
  int i = INDEX;
  hiprand_init(0, i, 0, &states[i]);
}

/*
Actually generates the estimate of pi.
*/
int main(int argc, char *argv[]) {
 
  int N,M;

  if (argc == 3) {
    N = 1 << atoi(argv[1]);
    M = 1 << atoi(argv[2]);
    printf("N: %d, M: %d\n",N, M);
  } else {
    N = NN;
    M = MM;
  }


  printf("sizeof hiprandState %d\n", sizeof(hiprandState));
  struct timeval begin, t1, t2; //, bs1, bs2;
  MARK_TIME(begin);
  printf("starting pi calc...\n");

  dim3 block, grid;
  block.x = THREADBLOCKSIZE;
  grid.x = (N + THREADBLOCKSIZE - 1)/THREADBLOCKSIZE;
  printf("grid.x %d\n", grid.x);
  printf("block.x %d\n", block.x);


  MARK_TIME(t1);
  printf("mallocing on host and device\n");
  float *p, *d_p;
  p = (float *)malloc(grid.x*sizeof(float));
  MARK_TIME(t2);
  printf("it took %f seconds to allocate p...\n", CALC_TIME(t1, t2));
  hipMalloc(&d_p,grid.x*sizeof(float));
  MARK_TIME(t1);
  printf("it took %f seconds to allocate d_p...\n", CALC_TIME(t2, t1));

  hiprandState *states;
  hipMalloc(&states, N*sizeof(hiprandState));
  MARK_TIME(t2);
  printf("it took %f seconds to allocate states...\n", CALC_TIME(t1, t2));

  printf("running kernel_setup...");
  MARK_TIME(t1);
  kernel_setup<<<grid, block>>>(states);
  MARK_TIME(t2);
  printf("done\n");
  printf("it took %f seconds to execute kernel_setup...\n", CALC_TIME(t1, t2));

  printf("running generate...");
  MARK_TIME(t1);
  generate<<<grid, block, block.x*sizeof(float)>>>(states, d_p, M);
  MARK_TIME(t2);
  printf("done\n");
  printf("it took %f seconds to execute generate...\n", CALC_TIME(t1, t2));

  printf("starting cuda memcpy...\n");
  MARK_TIME(t1);
  hipMemcpy(p, d_p, grid.x*sizeof(float), hipMemcpyDeviceToHost);
  MARK_TIME(t2);
  printf("done\n");
  printf("it took %f seconds to memcpy to host...\n", CALC_TIME(t1, t2));

  MARK_TIME(t1);
  int num_print = grid.x;
  float total = 0.0;
  for (int i = 0; i < num_print; i++) {
    //printf("i:%d\tsum %f\n",i,p[i]);
    total += p[i];
  }

  float pi = 4.0 * total / grid.x;
  printf("pi estimate: %f\n", pi);

  printf("cleaning up\n");
  hipFree(states);
  hipFree(d_p);
  free(p);
  MARK_TIME(t2);
  printf("it took %f seconds to calc total and clean up\n", CALC_TIME(t1,t2));
  printf("\nThe total execution time of this program was %f seconds\n", CALC_TIME(begin,t2));

  FILE *fp = fopen("gpu_results.dat", "a");
  fprintf(fp, "%d %d %.10f\n",N,M,pi);
  fclose(fp);

  return 0;
}
