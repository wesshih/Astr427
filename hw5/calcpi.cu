
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/time.h>

#define NN (1<<5) // number of seeds
#define MM (1<<10) // number of samples per seed
#define THREADBLOCKSIZE 1024
#define LENGTH (N*sizeof(float))
#define INDEX (blockIdx.x * blockDim.x + threadIdx.x)
#define MARK_TIME(t) gettimeofday(&t, NULL)
#define CALC_TIME(t1, t2) (1.0e6 * (t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec))/(1.0e6)
#define D2H hipMemcpyDeviceToHost
#define H2D hipMemcpyHostToDevice

typedef struct {
  float x, y;
} point;

extern __shared__ float sdat[];

__device__ void d_gen(hiprandState *globalState, int m) {
  int i = INDEX;
  hiprandState localState = globalState[i];

  int spt = m; //M; //number of samples per thread
  int tid = threadIdx.x;
  //int bid = blockIdx.x;
  int count = 0;
  //printf("tid:%d\tbid:%d\ti:%d\n",tid,bid,i);
  for (int s = 0; s < spt; s++) {
    float rx = hiprand_uniform(&localState);
    float ry = hiprand_uniform(&localState);
    float mag = rx*rx + ry*ry;
    if (mag <= 1.0f) {
      count += 1;
      sdat[tid] += 1.0;
    }
  }

  globalState[i] = localState;
  //sdat[tid] *= 4.0f;
  sdat[tid] *= 1.0/spt;

}

__device__ void d_count(float *sums) {
  int tid = threadIdx.x;
  for (int i = blockDim.x/2; i > 0; i >>= 1) {
    if (tid < i)
      sdat[tid] += sdat[tid + i];
   __syncthreads();
  }
  __syncthreads();
  if (tid == 0) {
    sdat[0] *= 1.0/blockDim.x;
    sums[blockIdx.x] = sdat[0];
  }
}

__global__ void generate(hiprandState *globalState, float *sums, int m) {
  d_gen(globalState, m);
  __syncthreads();
  d_count(sums);
  __syncthreads();
}

__global__ void kernel_setup(hiprandState *states) {
  int i = INDEX;
  hiprand_init(0, i, 0, &states[i]);
}

int main(int argc, char *argv[]) {
 
  int N,M;

  if (argc > 1) {
    if (argc != 3) {
      printf("wrong number of args. exiting\n");
      return -1;
    }
    N = 1 << atoi(argv[1]);
    M = 1 << atoi(argv[2]);
    printf("N: %d, M: %d\n",N, M);
  }


  printf("sizeof hiprandState %d\n", sizeof(hiprandState));
  struct timeval begin, t1, t2; //, bs1, bs2;
  MARK_TIME(begin);
  printf("starting pi calc...\n");

  dim3 block, grid;
  block.x = THREADBLOCKSIZE;
  grid.x = (N + THREADBLOCKSIZE - 1)/THREADBLOCKSIZE;
  printf("grid.x %d\n", grid.x);
  printf("block.x %d\n", block.x);


  MARK_TIME(t1);
  printf("mallocing on host and device\n");
  float *p, *d_p;
  p = (float *)malloc(grid.x*sizeof(float));
  MARK_TIME(t2);
  printf("it took %f seconds to allocate p...\n", CALC_TIME(t1, t2));
  hipMalloc(&d_p,grid.x*sizeof(float));
  MARK_TIME(t1);
  printf("it took %f seconds to allocate d_p...\n", CALC_TIME(t2, t1));

  hiprandState *states;
  hipMalloc(&states, N*sizeof(hiprandState));
  MARK_TIME(t2);
  printf("it took %f seconds to allocate states...\n", CALC_TIME(t1, t2));

  printf("running kernel_setup...");
  MARK_TIME(t1);
  kernel_setup<<<grid, block>>>(states);
  MARK_TIME(t2);
  printf("done\n");
  printf("it took %f seconds to execute kernel_setup...\n", CALC_TIME(t1, t2));

  printf("running generate...");
  MARK_TIME(t1);
  generate<<<grid, block, block.x*sizeof(float)>>>(states, d_p, M);
  MARK_TIME(t2);
  printf("done\n");
  printf("it took %f seconds to execute generate...\n", CALC_TIME(t1, t2));

  printf("starting cuda memcpy...\n");
  MARK_TIME(t1);
  hipMemcpy(p, d_p, grid.x*sizeof(float), hipMemcpyDeviceToHost);
  MARK_TIME(t2);
  printf("done\n");
  printf("it took %f seconds to memcpy to host...\n", CALC_TIME(t1, t2));

  MARK_TIME(t1);
  int num_print = grid.x;
  float total = 0.0;
  for (int i = 0; i < num_print; i++) {
    //printf("i:%d\tsum %f\n",i,p[i]);
    total += p[i];
  }

  float pi = 4.0 * total / grid.x;

  printf("inside: %f\n", total);
  printf("ratio: %f\n", 1.0f*total/grid.x);
  printf("pi estimate: %f\n", pi);

  printf("sizeof(float) %d\n",sizeof(float));
  printf("block (x,y,z): (%d,%d,%d)\n",block.x,block.y,block.z);
  printf("grid  (x,y,z): (%d,%d,%d)\n",grid.x, grid.y, grid.z);

  printf("cleaning up\n");
  hipFree(states);
  hipFree(d_p);
  free(p);
  MARK_TIME(t2);
  printf("it took %f seconds to calc total and clean up\n", CALC_TIME(t1,t2));
  printf("\nThe total execution time of this program was %f seconds\n", CALC_TIME(begin,t2));

FILE *fp = fopen("test.dat", "a");
//fseek(fp, SEEK_END);
fprintf(fp, "%d %d %.10f\n",N,M,pi);
fclose(fp);

}
