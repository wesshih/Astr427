
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/time.h>
//#include "multipi.h"

#define N (1<<10)
#define M (1<<10)
#define THREADBLOCKSIZE 1024
#define LENGTH (N*sizeof(point))
#define INDEX (blockIdx.x * blockDim.x + threadIdx.x)
#define D2H hipMemcpyDeviceToHost
#define H2D hipMemcpyHostToDevice
#define MARK_TIME(t) gettimeofday(&t, NULL)
#define CALC_TIME(t1, t2) (1.0e6 * (t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec))/(1.0e6)
#define PRINT_TIME(t, s) printf("It took %f seconds to do %s\n",t,s)


typedef struct {
  int num;
  float pi;
} CalcTest;

#define DEBUG False

extern __shared__ float sdat[];

// setup all the threads on a gpu with individual rand seeds
__global__ void
kernel_setup(hiprandState *states, int d) {
  int i = INDEX;
  hiprand_init(d, i, 0, &states[i]);
}

// generate M random points for each thread, and count how many have len <= 1.0
__device__ void
d_gen(hiprandState *globalState) {
  int i = INDEX;
  hiprandState localState = globalState[i];
  int tid = threadIdx.x;
  for (int s = 0; s < M; s++) {
    float rx = hiprand_uniform(&localState);
    float ry = hiprand_uniform(&localState);
    float mag = rx*rx + ry*ry;
    if (mag <= 1.0f) {
      sdat[tid] += 1.0;
    }
  }
  globalState[i] = localState;
  sdat[tid] *= 4.0f;
  sdat[tid] *= 1.0/M;
}

__device__ void
d_count(float *sums) {
  int tid = threadIdx.x;
  for (int i = blockDim.x/2; i > 0; i >>= 1) {
    if (tid < i)
      sdat[tid] += sdat[tid + i];
   __syncthreads();
  }
  __syncthreads();
  if (tid == 0) {
    sdat[0] *= 1.0/blockDim.x;
    sums[blockIdx.x] = sdat[0];
  }
}

// gets the points within a circle of radius 1.0
__global__ void
generate(hiprandState *globalState, float *sums) {
  d_gen(globalState);
  __syncthreads();
  d_count(sums);
}

CalcTest *
runTest(int num) {
  printf("calculating pi with num %d\n", num);
  CalcTest *t = (CalcTest *)malloc(sizeof(CalcTest));
  t->num = num;

  //printf("MultiGPU Pi is running...\n");
  struct timeval begin, t1, t2;
  //MARK_TIME(begin);

  //MARK_TIME(t1);
  int numDevs = 0;
  hipGetDeviceCount(&numDevs);
  numDevs = 1;
  //MARK_TIME(t2);
  //PRINT_TIME(CALC_TIME(t1,t2),"get device count");
  //printf("We have access to %d devices\n", numDevs);

  dim3 block, grid;
  block.x = THREADBLOCKSIZE;
  grid.x = (num + THREADBLOCKSIZE - 1)/THREADBLOCKSIZE;
  //printf("grid.x %d\n", grid.x);
  //printf("block.x %d\n", block.x);

  //MARK_TIME(t1);
  float *a[numDevs];
  for (int d = 0; d < numDevs; d++) {
    a[d] = (float *)malloc(grid.x*sizeof(float));
  }
  //MARK_TIME(t2);
  //PRINT_TIME(CALC_TIME(t1,t2),"allocal host arrays");

  // allocate memory on each device
  //printf("allocating memory...");
  //MARK_TIME(t1);
  float *d_a[numDevs];
  hiprandState *states[numDevs];
  for (int d = 0; d < numDevs; d++) {
    hipSetDevice(d);
    hipMalloc(&d_a[d],grid.x*sizeof(float));
    hipMalloc(&states[d],num*sizeof(hiprandState));
  }
  //MARK_TIME(t2);
  //PRINT_TIME(CALC_TIME(t1,t2),"allocal cuda arrays");
  //printf("done\n");

  // run the kernel on each device
  //printf("\nrunning kernels...");
  //MARK_TIME(t1);
  for (int d = 0; d < numDevs; d++) {
    hipSetDevice(d);
    kernel_setup<<<grid, block>>>(states[d],d);
    generate<<<grid, block, block.x*sizeof(float)>>>(states[d], d_a[d]);
  }
  //MARK_TIME(t2);
  //PRINT_TIME(CALC_TIME(t1,t2),"execute kernels");
  //printf("done\n");

  // copy data back to host
  //MARK_TIME(t1);
  for (int d = 0; d < numDevs; d++) {
    hipSetDevice(d);
    hipMemcpy(a[d],d_a[d],grid.x*sizeof(float), D2H);
  }
  //MARK_TIME(t2);
  //PRINT_TIME(CALC_TIME(t1,t2),"copy mem back to host");

  // now print the host arrays
  //MARK_TIME(t1);
  int num_print = 2;//grid.x;
  float total = 0.0;
  for (int d = 0; d < numDevs; d++) {
    printf("Values received from device %d:\n",d);
    for (int i = 0; i < grid.x; i++) {
      total += a[d][i];
      if (i < num_print) printf("\ta[%d][%d]: %f\n",d,i,a[d][i]);
    }
  }

  total *= 1.0/(grid.x * numDevs);
  printf("Estimate of pi: %f\n",total);
  t->pi = total;

  for (int d = 0; d < numDevs; d++) {
    free(a[d]);
    hipFree(d_a[d]);
  }
  //MARK_TIME(t2);
  //PRINT_TIME(CALC_TIME(t1,t2),"print results and clean up");

  //PRINT_TIME(CALC_TIME(begin,t2),"execute the entire program");

  return t;
}

int
main(void) {

  /*
  printf("MultiGPU Pi is running...\n");
  struct timeval begin, t1, t2;
  MARK_TIME(begin);

  MARK_TIME(t1);
  int numDevs = 0;
  cudaGetDeviceCount(&numDevs);
  numDevs = 2;
  MARK_TIME(t2);
  PRINT_TIME(CALC_TIME(t1,t2),"get device count");
  printf("We have access to %d devices\n", numDevs);

  dim3 block, grid;
  block.x = THREADBLOCKSIZE;
  grid.x = (N + THREADBLOCKSIZE - 1)/THREADBLOCKSIZE;
  printf("grid.x %d\n", grid.x);
  printf("block.x %d\n", block.x);

  MARK_TIME(t1);
  float *a[numDevs];
  for (int d = 0; d < numDevs; d++) {
    a[d] = (float *)malloc(grid.x*sizeof(float));
  }
  MARK_TIME(t2);
  PRINT_TIME(CALC_TIME(t1,t2),"allocal host arrays");

  // allocate memory on each device
  printf("allocating memory...");
  MARK_TIME(t1);
  float *d_a[numDevs];
  curandState *states[numDevs];
  for (int d = 0; d < numDevs; d++) {
    cudaSetDevice(d);
    cudaMalloc(&d_a[d],grid.x*sizeof(float));
    cudaMalloc(&states[d],N*sizeof(curandState));
  }
  MARK_TIME(t2);
  PRINT_TIME(CALC_TIME(t1,t2),"allocal cuda arrays");
  printf("done\n");

  // run the kernel on each device
  printf("\nrunning kernels...");
  MARK_TIME(t1);
  for (int d = 0; d < numDevs; d++) {
    cudaSetDevice(d);
    kernel_setup<<<grid, block>>>(states[d],d);
    generate<<<grid, block, block.x*sizeof(float)>>>(states[d], d_a[d]);
  }
  MARK_TIME(t2);
  PRINT_TIME(CALC_TIME(t1,t2),"execute kernels");
  printf("done\n");

  // copy data back to host
  MARK_TIME(t1);
  for (int d = 0; d < numDevs; d++) {
    cudaSetDevice(d);
    cudaMemcpy(a[d],d_a[d],grid.x*sizeof(float), D2H);
  }
  MARK_TIME(t2);
  PRINT_TIME(CALC_TIME(t1,t2),"copy mem back to host");

  // now print the host arrays
  MARK_TIME(t1);
  int num_print = grid.x;
  float total = 0.0;
  for (int d = 0; d < numDevs; d++) {
    printf("Values received from device %d:\n",d);
    for (int i = 0; i < grid.x; i++) {
      total += a[d][i];
      if (i < num_print) printf("\ta[%d][%d]: %f\n",d,i,a[d][i]);
    }
  }

  total *= 1.0/(grid.x * numDevs);
  printf("Estimate of pi: %f\n",total);

  for (int d = 0; d < numDevs; d++) {
    free(a[d]);
    cudaFree(d_a[d]);
  }
  MARK_TIME(t2);
  PRINT_TIME(CALC_TIME(t1,t2),"print results and clean up");

  PRINT_TIME(CALC_TIME(begin,t2),"execute the entire program");
  */

  int numTests = 10;
  CalcTest *tests[numTests];
  for (int i = 0; i < numTests; i++) {
    tests[i] = runTest(N);
  }

  FILE *fp;
  fp = fopen("results_multi.txt","w");
  fprintf(fp, "N\tpi\n");
  for (int i = 0; i < numTests; i++) {
    fprintf(fp, "%d\t%f\n", tests[i]->num, tests[i]->pi);
  }
  fclose(fp);

  return 0;
}

